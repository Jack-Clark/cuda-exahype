/* 
	@author Jack Clark

	Simple program to simulate 2D advection using the finite volume approach, with naive averaging at cell boundaries. 
	
	Compile with nvcc -O3 advection.cu -o gpu_advection
*/


#include <fstream>
#include <sstream>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define NUM_CELLS_X 40
#define NUM_CELLS_Y 40
#define TIMESTEP 0.001
#define NUM_TIMESTEPS 100000
#define DELTA_X 1
#define DELTA_Y 1
#define PLOT_FREQUENCY 100

// CPU data
double h_flux_x[NUM_CELLS_X*NUM_CELLS_Y];
double h_flux_y[NUM_CELLS_X*NUM_CELLS_Y];
double h_q[NUM_CELLS_X*NUM_CELLS_Y];
double h_velocities[2];
double h_max_velocity;

// GPU data
double * d_flux_x;
double * d_flux_y;
double * d_q;
double * d_max_velocity;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int getIndex(int row, int col) {
	return row * NUM_CELLS_X + col;
}

void printCSVFile(int counter) {
  std::stringstream filename;
  filename << "result-" << counter <<  ".csv";
  std::ofstream out( filename.str().c_str() );

  out << "x, y, z" << std::endl;

  for (int i=0; i<NUM_CELLS_Y; i++) {
  	for(int j=0; j<NUM_CELLS_X; j++) {
	    out << i
	        << ","
	        << j
	 	    << ","
	        << h_q[i*NUM_CELLS_X+j]
	        << std::endl;
	}
  }
}

void printResult(int timestep) {

	printf("\nTimestep %d \n", timestep);
	printf("Q: ");
	for (int i=0; i<NUM_CELLS_Y; i++) {
		printf("\n");
  		for(int j=0; j<NUM_CELLS_X; j++) {
  			printf("%f ", h_q[getIndex(i,j)]);
  		}
  	}
	printf("\n");
	printf("Flux_X: ");
	for (int i=0; i<NUM_CELLS_Y; i++) {
		printf("\n");
  		for(int j=0; j<NUM_CELLS_X; j++) {
  			printf("%f ", h_flux_x[getIndex(i,j)]);
  		}
  	}
	printf("\n");
	printf("Flux_Y: ");
	for (int i=0; i<NUM_CELLS_Y; i++) {
		printf("\n");
  		for(int j=0; j<NUM_CELLS_X; j++) {
  			printf("%f ", h_flux_y[getIndex(i,j)]);
  		}
  	}
	printf("\n");
}

void setup() {
	for (int i=1; i<4; i++) {
  		for(int j=1; j<4; j++) {
  			int index = getIndex(i,j);
  			h_q[index] = 5.0;
  		}
  	}
	h_velocities[0] = 0.5;
	h_velocities[1] = 0.5;
	h_max_velocity = 0.0;
	int i;
	for(i = 0; i<sizeof(h_velocities)/sizeof(double); i++) {
		h_max_velocity += h_velocities[i] * h_velocities[i];
	}
	h_max_velocity = sqrt(h_max_velocity);
}

void reconstruction() {
	for(int i=0; i<NUM_CELLS_Y; i++) {
		for(int j=0; j<NUM_CELLS_X; j++) {
			int index = getIndex(i,j);
			if(j != 0) {
				h_flux_x[index] = ((h_q[index] + h_q[index-1]) / 2) - (h_max_velocity/2 * (h_q[index] - h_q[index-1]));
			}
			if(i != 0) {
				h_flux_y[index] = ((h_q[index] + h_q[index-NUM_CELLS_X]) / 2) - (h_max_velocity/2 * (h_q[index] - h_q[index-NUM_CELLS_X]));
			}
		}
	}
}

__global__ void reconstruction_kernel(const double * const d_q, double * d_flux_x, double * d_flux_y, const double * const d_max_velocity) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= NUM_CELLS_Y || col >= NUM_CELLS_X)
		return;

	int self = row * NUM_CELLS_X + col;

	if(col != 0) {
		d_flux_x[self] = ((d_q[self] + d_q[self-1]) / 2) - (*d_max_velocity/2 * (d_q[self] - d_q[self-1]));
	}

	if(row != 0) {
		d_flux_y[self] = ((d_q[self] + d_q[self-NUM_CELLS_X]) / 2) - (*d_max_velocity/2 * (d_q[self] - d_q[self-NUM_CELLS_X]));
	}
}

__global__ void update_kernel(double * d_q, const double * const d_flux_x, const double * const d_flux_y) {  
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= NUM_CELLS_Y || col >= NUM_CELLS_X)
		return;

	int self = row * NUM_CELLS_X + col;

	if(self > ((NUM_CELLS_X-1) * (NUM_CELLS_Y-1) - 1))
		return;

	double temp = d_q[self] + ((TIMESTEP/DELTA_X) * (d_flux_x[self] - d_flux_x[self+1]));

	if(row < NUM_CELLS_X-1)
		temp += (TIMESTEP/DELTA_Y) * (d_flux_y[self] - d_flux_y[self+NUM_CELLS_X]);

	// 0 limiter to remove numerical artefacts 
	if(temp <= 0.0) {
		d_q[self] = 0.0;
	} else {
		d_q[self] = temp;
	}

}

void update_cells() {
	double temp;
	for(int i=0; i<NUM_CELLS_Y-1; i++) {
		for(int j=0; j<NUM_CELLS_X-1; j++) {
			int index = getIndex(i,j);
			assert(index >= 0);
			assert(index < NUM_CELLS_X * NUM_CELLS_Y);
			temp = h_q[index] + ((TIMESTEP/DELTA_X) * (h_flux_x[index] - h_flux_x[index+1])) + ((TIMESTEP/DELTA_Y) * (h_flux_y[index] - h_flux_y[index+NUM_CELLS_X]));
			if(temp < 0) {
				h_q[index] = 0;
			} else {
				h_q[index] = temp;
			}
		}
	}
}


int main() {
	setup();
	printCSVFile(0);
	//printResult(0);

	// Allocate memory for GPU data - Naive assumption that our GPU can fit all our data in memory at once
	gpuErrchk(hipMalloc((void **) &d_flux_x, sizeof(double) * NUM_CELLS_X * NUM_CELLS_Y));
	gpuErrchk(hipMalloc((void **) &d_flux_y, sizeof(double) * NUM_CELLS_X * NUM_CELLS_Y));
	gpuErrchk(hipMalloc((void **) &d_q, sizeof(double) * NUM_CELLS_X * NUM_CELLS_Y));
	gpuErrchk(hipMalloc((void **) &d_max_velocity, sizeof(double)));

	gpuErrchk(hipMemset(d_flux_x, 0, sizeof(double) * NUM_CELLS_X * NUM_CELLS_Y));
	gpuErrchk(hipMemset(d_flux_y, 0, sizeof(double) * NUM_CELLS_X * NUM_CELLS_Y));
	gpuErrchk(hipMemcpy(d_max_velocity, &h_max_velocity, sizeof(double), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_q, h_q, sizeof(double) * NUM_CELLS_X * NUM_CELLS_Y, hipMemcpyHostToDevice));
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	dim3 block(32, 32);

	for(int i=0; i<NUM_TIMESTEPS; i++) {
		reconstruction_kernel<<<3, block>>>(d_q, d_flux_x, d_flux_y, d_max_velocity);
		gpuErrchk( hipPeekAtLastError() );
		update_kernel<<<3, block>>>(d_q, d_flux_x, d_flux_y);
		gpuErrchk( hipPeekAtLastError() );

		// Copy GPU data back to GPU for recording - TODO: Improve this so that we only copy data back when we need it.
		gpuErrchk(hipMemcpy(h_q, d_q, sizeof(double) * NUM_CELLS_X * NUM_CELLS_Y, hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(h_flux_x, d_flux_x, sizeof(double) * NUM_CELLS_X * NUM_CELLS_Y, hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(h_flux_y, d_flux_y, sizeof(double) * NUM_CELLS_X * NUM_CELLS_Y, hipMemcpyDeviceToHost));

		if (i%PLOT_FREQUENCY==0) {
      		printCSVFile(i/PLOT_FREQUENCY+1); // Please switch off all IO if you do performance tests.
      		//printResult(i+1);
    	}
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf ("Time for the program: %f ms\n", time);

	hipFree(d_flux_x);
	hipFree(d_flux_y);
	hipFree(d_q);
	hipFree(d_max_velocity);

	return 0;
}