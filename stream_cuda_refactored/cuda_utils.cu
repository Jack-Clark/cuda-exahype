
#include <hip/hip_runtime.h>
#include <stdio.h>

void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__device__ int idx(int threadIdxX, int threadIdxY, int blockIdxX, int numVariables, int basisSize, int patchBegin, int idxSelector) {
  switch(idxSelector) {
    case 0: // This is the idx(j, k, 0) index for the x face
      if(threadIdx.x < patchBegin) {
        return -1;
      }
      return threadIdx.x * (numVariables * (basisSize+2)) + (threadIdx.y * numVariables);
      break;

    case 1: // This is the idx(j, k+1, 0) index for the x face
      if(threadIdx.x < patchBegin) {
        return -1;
      }
      return threadIdx.x * (numVariables * (basisSize+2)) + ((threadIdx.y+1) * numVariables);
      break;
    
    case 2: // This is the idx(j, k, 0) index for the y edge
      if(threadIdx.y < patchBegin) {
        return -1;
      }
      return threadIdx.x * (numVariables * (basisSize+2)) + (threadIdx.y * numVariables);
      break;

    case 3: // This is the idx(j+1, k, 0) index for the y edge
      if(threadIdx.y < patchBegin) {
        return -1;
      }
      return (threadIdx.x+1) * (numVariables * (basisSize+2)) + (threadIdx.y * numVariables);
      break;

    default:
      return -1;
      break;
  }
}