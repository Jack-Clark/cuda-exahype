#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "cuda_utils.cuh"

/* 
 * The user's eigenvalues CUDA implementation should go here. All that is required to change from a CPU version to a CUDA version
 * is for the user to add the value of q_arr_idx to their Q array indices and to add other_idx to all other array indices. For example:
 * Q[1] becomes Q[q_arr_idx + 1] and lambda[1] becomes lambda[other_idx + 1].
 */
__global__ void eigenvalues_kernel(const double* const Q, const int normalNonZeroIndex, double* lambda, const int numVariables, const int patchBegin, const int basisSize, const int idxSelector) {
  // Dimensions             = 2
  // Number of variables    = 5 (#unknowns + #parameters)
  const int other_idx = threadIdx.x + blockIdx.x * threadIdx.y * numVariables;
  const int q_arr_idx = idx(threadIdx.x, threadIdx.y, blockIdx.x, numVariables, basisSize, patchBegin, idxSelector);
  if(q_arr_idx == -1) 
    return;

  // Application code goes here
  const double GAMMA = 1.4;

  const double irho = 1.0/Q[q_arr_idx];
  const double p = (GAMMA-1) * (Q[q_arr_idx+4] - 0.5 * (Q[q_arr_idx+1] * Q[q_arr_idx+1] + Q[q_arr_idx+2] * Q[q_arr_idx+2]) * irho);   

  const double u_n = Q[q_arr_idx+normalNonZeroIndex + 1] * irho;
  const double c = sqrt(GAMMA * p * irho);

  lambda[other_idx] = u_n - c;
  lambda[other_idx+1] = u_n;
  lambda[other_idx+2] = u_n;
  lambda[other_idx+3] = u_n;
  lambda[other_idx+4] = u_n + c;
}

/* 
 * The user's flux CUDA implementation should go here. All that is required to change from a CPU version to a CUDA version
 * is for the user to add the value of q_arr_idx to their Q array indices and to add other_idx to all other array indices. For example:
 * Q[1] becomes Q[q_arr_idx + 1] and f[1] becomes f[other_idx + 1].
 */
__global__ void flux_kernel(const double* const Q, double** F, const int numVariables, const int patchBegin, const int basisSize, const int idxSelector) {
  // Dimensions             = 2
  // Number of variables    = 5 (#unknowns + #parameters)
  const int other_idx = threadIdx.x + blockIdx.x * threadIdx.y * numVariables;
  const int q_arr_idx = idx(threadIdx.x, threadIdx.y, blockIdx.x, numVariables, basisSize, patchBegin, idxSelector);
  if(q_arr_idx == -1) 
    return;

  const double GAMMA = 1.4;

  const double irho = 1.0/Q[q_arr_idx];
  const double p = (GAMMA-1) * (Q[q_arr_idx+4] - 0.5 * (Q[q_arr_idx+1] * Q[q_arr_idx+1] + Q[q_arr_idx+2] * Q[q_arr_idx+2]) * irho);

  double* f = F[0];
  double* g = F[1];

  f[other_idx] = Q[q_arr_idx+1]; // should be numVariables * tid
  f[other_idx+1] = irho * Q[q_arr_idx+1] * Q[q_arr_idx+1] + p;
  f[other_idx+2] = irho * Q[q_arr_idx+1] * Q[q_arr_idx+2];
  f[other_idx+3] = irho * Q[q_arr_idx+1] * Q[q_arr_idx+3];
  f[other_idx+4] = irho * Q[q_arr_idx+1] * (Q[q_arr_idx+4] + p);

  g[other_idx] = Q[q_arr_idx+2]; // Should be numVariables * tid + numVariables
  g[other_idx+1] = irho * Q[q_arr_idx+2] * Q[q_arr_idx+1];
  g[other_idx+2] = irho * Q[q_arr_idx+2] * Q[q_arr_idx+2] + p;
  g[other_idx+3] = irho * Q[q_arr_idx+2] * Q[q_arr_idx+3];
  g[other_idx+4] = irho * Q[q_arr_idx+2] * (Q[q_arr_idx+4] + p);
}