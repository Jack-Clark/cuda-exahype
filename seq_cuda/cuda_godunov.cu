#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <string.h>
#include "kernels/KernelUtils.h"
#include "tarch/la/Vector.h"

#define DIMENSIONS 2


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void gen_eigenvalues_kernel(const double* const Q, const int normalNonZeroIndex, double* lambda, const int numVariables, const int patchBegin, const int basisSize, const int idxSelector) {
  // Dimensions             = 2
  // Number of variables    = 5 (#unknowns + #parameters)
  const int ltid = threadIdx.x + blockIdx.x * threadIdx.y * numVariables;
  int gtid;
  switch(idxSelector) {
    case 0: // This is the idx(j, k, 0) index for the x face
      if(threadIdx.x < patchBegin) {
        return;
      }
      gtid = threadIdx.x * (numVariables * (basisSize+2)) + (threadIdx.y * numVariables);
      break;

    case 1: // This is the idx(j, k+1, 0) index for the x face
      if(threadIdx.x < patchBegin) {
        return;
      }
      gtid = threadIdx.x * (numVariables * (basisSize+2)) + ((threadIdx.y+1) * numVariables);
      break;
    
    case 2: // This is the idx(j, k, 0) index for the y edge
      if(threadIdx.y < patchBegin) {
        return;
      }
      gtid = threadIdx.x * (numVariables * (basisSize+2)) + (threadIdx.y * numVariables);
      break;

    case 3: // This is the idx(j+1, k, 0) index for the y edge
      if(threadIdx.y < patchBegin) {
        return;
      }
      gtid = (threadIdx.x+1) * (numVariables * (basisSize+2)) + (threadIdx.y * numVariables);
      break;

    default:
      gtid = 0;
      assert(false);
      break;
  }

  // Application code goes here
  const double GAMMA = 1.4;

  const double irho = 1.0/Q[gtid];
  const double p = (GAMMA-1) * (Q[gtid+4] - 0.5 * (Q[gtid+1] * Q[gtid+1] + Q[gtid+2] * Q[gtid+2]) * irho);   

  const double u_n = Q[gtid+normalNonZeroIndex + 1] * irho;
  const double c = sqrt(GAMMA * p * irho);

  lambda[ltid] = u_n - c;
  lambda[ltid+1] = u_n;
  lambda[ltid+2] = u_n;
  lambda[ltid+3] = u_n;
  lambda[ltid+4] = u_n + c;
}

__global__ void gen_flux_kernel_2D(const double* const Q, double** F, const int numVariables, const int patchBegin, const int basisSize, const int idxSelector) {
  // Dimensions             = 2
  // Number of variables    = 5 (#unknowns + #parameters)
  const int ltid = threadIdx.x + blockIdx.x * threadIdx.y * numVariables;
  int gtid = 0;
  switch(idxSelector) {
    case 0: // This is the idx(j, k, 0) index for the x face
      if(threadIdx.x < patchBegin) {
        return;
      }
      gtid = threadIdx.x * (numVariables * (basisSize+2)) + (threadIdx.y * numVariables);
      break;

    case 1: // This is the idx(j, k+1, 0) index for the x face
      if(threadIdx.x < patchBegin) {
        return;
      }
      gtid = threadIdx.x * (numVariables * (basisSize+2)) + ((threadIdx.y+1) * numVariables);
      break;
    
    case 2: // This is the idx(j, k, 0) index for the y edge
      if(threadIdx.y < patchBegin) {
        return;
      }
      gtid = threadIdx.x * (numVariables * (basisSize+2)) + (threadIdx.y * numVariables);
      break;

    case 3: // This is the idx(j+1, k, 0) index for the y edge
      if(threadIdx.y < patchBegin) {
        return;
      }
      gtid = (threadIdx.x+1) * (numVariables * (basisSize+2)) + (threadIdx.y * numVariables);
      break;

    default:
      gtid = 0;
      assert(false);
      break;
  }

  // Application code goes here
  const double GAMMA = 1.4;

  const double irho = 1.0/Q[gtid];
  const double p = (GAMMA-1) * (Q[gtid+4] - 0.5 * (Q[gtid+1] * Q[gtid+1] + Q[gtid+2] * Q[gtid+2]) * irho);

  double* f = F[0];
  double* g = F[1];

  f[ltid] = Q[gtid+1];
  f[ltid+1] = irho * Q[gtid+1] * Q[gtid+1] + p;
  f[ltid+2] = irho * Q[gtid+1] * Q[gtid+2];
  f[ltid+3] = irho * Q[gtid+1] * Q[gtid+3];
  f[ltid+4] = irho * Q[gtid+1] * (Q[gtid+4] + p);

  g[ltid] = Q[gtid+2];
  g[ltid+1] = irho * Q[gtid+2] * Q[gtid+1];
  g[ltid+2] = irho * Q[gtid+2] * Q[gtid+2] + p;
  g[ltid+3] = irho * Q[gtid+2] * Q[gtid+3];
  g[ltid+4] = irho * Q[gtid+2] * (Q[gtid+4] + p);
}


__global__ void updateF_kernel(const double* const Q, double** FL2, double** FR2, double* F, const double* d_maxes, const int numVariables, 
                               const int patchBegin, const int basisSize, const int normalNonZero, const int xFlag) {
  // Dimensions             = 2
  // Number of variables    = 5 (#unknowns + #parameters)

  const int ltid = threadIdx.x + blockIdx.x * threadIdx.y * numVariables;
  int qL_idx;
  int qR_idx;
  if(xFlag) {
      if(threadIdx.x < patchBegin) {
        return;
      }
      qL_idx = threadIdx.x * (numVariables * (basisSize+2)) + (threadIdx.y * numVariables);
      qR_idx = threadIdx.x * (numVariables * (basisSize+2)) + ((threadIdx.y+1) * numVariables);
  } else {
      if(threadIdx.y < patchBegin) {
        return;
      }
      qL_idx = threadIdx.x * (numVariables * (basisSize+2)) + (threadIdx.y * numVariables);
      qR_idx = (threadIdx.x+1) * (numVariables * (basisSize+2)) + (threadIdx.y * numVariables);
  }

  int tid = threadIdx.x + blockIdx.x * threadIdx.y;
  F[ltid]   = 0.5 * (FL2[normalNonZero][ltid]   + FR2[normalNonZero][ltid])   + 0.5 * d_maxes[tid] * (Q[qL_idx]   - Q[qR_idx]);
  F[ltid+1] = 0.5 * (FL2[normalNonZero][ltid+1] + FR2[normalNonZero][ltid+1]) + 0.5 * d_maxes[tid] * (Q[qL_idx+1] - Q[qR_idx+1]);
  F[ltid+2] = 0.5 * (FL2[normalNonZero][ltid+2] + FR2[normalNonZero][ltid+2]) + 0.5 * d_maxes[tid] * (Q[qL_idx+2] - Q[qR_idx+2]);
  F[ltid+3] = 0.5 * (FL2[normalNonZero][ltid+3] + FR2[normalNonZero][ltid+3]) + 0.5 * d_maxes[tid] * (Q[qL_idx+3] - Q[qR_idx+3]);
  F[ltid+4] = 0.5 * (FL2[normalNonZero][ltid+4] + FR2[normalNonZero][ltid+4]) + 0.5 * d_maxes[tid] * (Q[qL_idx+4] - Q[qR_idx+4]);
}

/* Finds the max value in sL and sR per cell and stores in global memory to be used later. */
__global__ void compute_maxes_kernel(double* sL, double* sR, double* maxes, const int numVariables) {
  
  const int ltid = threadIdx.x + blockIdx.x * threadIdx.y * numVariables;

  double max = -1.0;
  for(int i = 0; i < numVariables; i++) {
    const double abs_sL = fabs(sL[ltid+i]);
    const double abs_sR = fabs(sR[ltid+i]);
    const double tmp_max = fmax(abs_sL, abs_sR);
    max = fmax(max, tmp_max);
  }
  maxes[threadIdx.x + blockIdx.x * threadIdx.y] = max;

}

int c_index(int i, int j, int k, int basis, int numVariables) {
  return i * (basis * numVariables) + j * numVariables + k;
}

/**
 * Solves all the Riemann problems that do only require
 * internal data and add the result directly onto the
 * new solution. 
 * Finally add the source terms.
 */
extern "C"
double cudaSolutionUpdate(double* luh_new, const double* luh, 
                          int numberOfVariables, int basisSize,
                          double cflFactor, double dt_max_allowed,
                          const double cellSize[], int patchBegin,
                          int patchEnd, double dt) { 

  // Start CUDA
  int numCudaThreads = (patchEnd+1) * (patchEnd+1);
  int cudaMemReq = c_index(patchEnd+1, patchEnd, 5, basisSize+2, numberOfVariables);

  // Allocate and Transfer Memory
  double * d_Fn;
  double * d_sL;
  double * d_sR;
  double ** d_FL2;
  double ** d_FR2;
  double * d_q;
  gpuErrchk( hipMalloc((void **) &d_Fn, numCudaThreads * numberOfVariables * sizeof(double)) );
  gpuErrchk( hipMalloc((void **) &d_sL, numCudaThreads * numberOfVariables * sizeof(double)) );
  gpuErrchk( hipMalloc((void **) &d_sR, numCudaThreads * numberOfVariables * sizeof(double)) );
  gpuErrchk( hipMalloc((void **) &d_q, cudaMemReq * sizeof(double)) );

  gpuErrchk( hipMemcpy(d_q, luh, cudaMemReq * sizeof(double), hipMemcpyHostToDevice) );

  dim3 dimBlock(patchEnd+1, patchEnd+1);
  int normalNonZeroIndex = 0;

  // Compute x edges
  gen_eigenvalues_kernel<<<1, dimBlock>>>(d_q, normalNonZeroIndex, d_sL, numberOfVariables, patchBegin, basisSize, 0);
  gpuErrchk( hipPeekAtLastError() );
  gen_eigenvalues_kernel<<<1, dimBlock>>>(d_q, normalNonZeroIndex, d_sR, numberOfVariables, patchBegin, basisSize, 1);
  gpuErrchk( hipPeekAtLastError() );

  double * d_maxes;
  gpuErrchk( hipMalloc((void **) &d_maxes, sizeof(double) * numberOfVariables * numCudaThreads) );
  hipMemset(d_maxes, 0, sizeof(double) * numberOfVariables * numCudaThreads);

  compute_maxes_kernel<<<1, dimBlock>>>(d_sL, d_sR, d_maxes, numberOfVariables);
  gpuErrchk( hipPeekAtLastError() );

  double maxes[numCudaThreads];
  memset(maxes, 0, sizeof(double) * numCudaThreads);

  gpuErrchk( hipMemcpy(maxes, d_maxes, numCudaThreads * sizeof(double), hipMemcpyDeviceToHost) );

  double s_max = -1.0;
  for (int j = 0; j < numCudaThreads; j++) {
    const double abs_temp = fabs(maxes[j]);
    s_max = fmax( abs_temp, s_max );
  }

  // TODO(guera): Improve. I'm quite sure this is not the correct/best
  // formula. TODO(Dominic): The division by DIMENSIONS might make sure that C_x+C_y < 1
  dt_max_allowed = fmin(
      dt_max_allowed, cflFactor / DIMENSIONS * cellSize[0] / s_max); // TODO(Dominic): Ignore this for a while
  double dt_max_1 = dt_max_allowed;

  int numDimensions = 2;
  /* This next section takes a bit of getting used to if you're not familiar with GPU programming.
     Basically, we create an array in host memory of device pointers that have been allocated device memory.
     This is necessary in order to work with pointer->pointer types in CUDA.
  */
  gpuErrchk( hipMalloc((void **) &d_FL2, numDimensions * sizeof(double *)) );
  double* devicePointersStoredInHostMemoryL[numDimensions]; 
  for(int i = 0; i < numDimensions; i++) {
      gpuErrchk( hipMalloc( (void**)&devicePointersStoredInHostMemoryL[i], numCudaThreads * numberOfVariables * sizeof(double)) );
  }
  gpuErrchk( hipMemcpy(d_FL2, devicePointersStoredInHostMemoryL, sizeof(double*) * numDimensions, hipMemcpyHostToDevice) );

  gpuErrchk( hipMalloc((void **) &d_FR2, numDimensions * sizeof(double *)) );
  double* devicePointersStoredInHostMemoryR[numDimensions]; 
  for(int i = 0; i < numDimensions; i++) {
      gpuErrchk( hipMalloc( (void**)&devicePointersStoredInHostMemoryR[i], numCudaThreads * numberOfVariables * sizeof(double)) );
  }
  gpuErrchk( hipMemcpy(d_FR2, devicePointersStoredInHostMemoryR, sizeof(double*) * numDimensions, hipMemcpyHostToDevice) );

  gen_flux_kernel_2D<<<1, dimBlock>>>(d_q, d_FL2, numberOfVariables, patchBegin, basisSize, 0);
  gpuErrchk( hipPeekAtLastError() );
  gen_flux_kernel_2D<<<1, dimBlock>>>(d_q, d_FR2, numberOfVariables, patchBegin, basisSize, 1);
  gpuErrchk( hipPeekAtLastError() );

  updateF_kernel<<<1, dimBlock>>>(d_q, d_FL2, d_FR2, d_Fn, d_maxes, numberOfVariables, patchBegin, basisSize, normalNonZeroIndex, 1);
  gpuErrchk( hipPeekAtLastError() );

  double Fn[numCudaThreads * numberOfVariables];
  gpuErrchk( hipMemcpy(Fn, d_Fn, numCudaThreads * numberOfVariables * sizeof(double), hipMemcpyDeviceToHost) );
  
  for (int j = patchBegin; j < patchEnd+1; j++) {
    for (int k = patchBegin-1; k < patchEnd+1; k++) {
      for (int l=0; l<numberOfVariables; ++l) {
        luh_new[c_index(j, k, l, basisSize+2, numberOfVariables)]   -= dt / cellSize[0] * Fn[l];  
        luh_new[c_index(j, k+1, l, basisSize+2, numberOfVariables)] += dt / cellSize[0] * Fn[l];
      }
    }
  }

  // Compute y faces
  normalNonZeroIndex = 1;
  gen_eigenvalues_kernel<<<1, dimBlock>>>(d_q, normalNonZeroIndex, d_sL, numberOfVariables, patchBegin, basisSize, 2);
  gpuErrchk( hipPeekAtLastError() );
  gen_eigenvalues_kernel<<<1, dimBlock>>>(d_q, normalNonZeroIndex, d_sR, numberOfVariables, patchBegin, basisSize, 3);
  gpuErrchk( hipPeekAtLastError() );

  compute_maxes_kernel<<<1, dimBlock>>>(d_sL, d_sR, d_maxes, numberOfVariables);
  gpuErrchk( hipPeekAtLastError() );

  gpuErrchk( hipMemcpy(maxes, d_maxes, numCudaThreads * sizeof(double), hipMemcpyDeviceToHost) );

  s_max = -1.0;
  for (int j = 0; j < (patchEnd+1)*(patchEnd+1); j++) {
    const double abs_temp = fabs(maxes[j]);
    //printf("max[%d] = %f\n", j, abs_temp);
    s_max = fmax( abs_temp, s_max );
  }

  // TODO(guera): Improve. I'm quite sure this is not the correct/best
  // formula. TODO(Dominic): The division by DIMENSIONS might make sure that C_x+C_y < 1
  dt_max_allowed = fmin(
      dt_max_allowed, cflFactor / DIMENSIONS * cellSize[1] / s_max); // TODO(Dominic): Ignore this for a while
  double dt_max_2 = dt_max_allowed;

  gen_flux_kernel_2D<<<1, dimBlock>>>(d_q, d_FL2, numberOfVariables, patchBegin, basisSize, 2);
  gpuErrchk( hipPeekAtLastError() );
  gen_flux_kernel_2D<<<1, dimBlock>>>(d_q, d_FR2, numberOfVariables, patchBegin, basisSize, 3);
  gpuErrchk( hipPeekAtLastError() );

  updateF_kernel<<<1, dimBlock>>>(d_q, d_FL2, d_FR2, d_Fn, d_maxes, numberOfVariables, patchBegin, basisSize, normalNonZeroIndex, 0);
  gpuErrchk( hipPeekAtLastError() );

  gpuErrchk( hipMemcpy(Fn, d_Fn, numCudaThreads * numberOfVariables * sizeof(double), hipMemcpyDeviceToHost) );
  
  for (int j = patchBegin-1; j < patchEnd+1; j++) {
    for (int k = patchBegin; k < patchEnd+1; k++) {
      for (int l=0; l<numberOfVariables; ++l) {
        luh_new[c_index(j, k, l, basisSize+2, numberOfVariables)]   -= dt / cellSize[1] * Fn[l];  
        luh_new[c_index(j+1, k, l, basisSize+2, numberOfVariables)] += dt / cellSize[1] * Fn[l];
      }
    }
  }

  hipFree(d_Fn);
  hipFree(d_sL);
  hipFree(d_sR);
  hipFree(d_maxes);
  hipFree(d_q);
  for(int i = 0; i < DIMENSIONS; i++) {
      hipFree(devicePointersStoredInHostMemoryL[i]);
      hipFree(devicePointersStoredInHostMemoryR[i]);
  }
  hipFree(d_FR2);
  hipFree(d_FL2);
    
  return dt_max_allowed;
}
